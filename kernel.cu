
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


__global__ void add(int *a,  int *b,  int *c,int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void random_ints(int* x, int size)
{
    int i;
    for (i = 0; i < size; i++) {
        x[i] = rand() % 10;
    }
}

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512
int main()
{
    int  *a, *b, *c; //host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);

    //Allocate space for device copies a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    //Alloc space for host copies of a,b,c and setup input values
    a = (int *)malloc(size);  random_ints(a, N);
    b = (int *)malloc(size);  random_ints(b, N);
    c = (int *)malloc(size); 

  

    //Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, b, size, hipMemcpyHostToDevice);

    //Launch add() kernel on GPU with N blocks
    add<<<(N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    //Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    //Clean up
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);



}
